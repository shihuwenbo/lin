#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// allocate memory on gpu
extern "C" void cu_safe_malloc(float **g_f, size_t n_elem,
        size_t sizeof_elem) {
    void *gptr;
    hipError_t crc = hipMalloc(&gptr, n_elem*sizeof_elem);
    if(crc) {
        printf("hipMalloc Error=%d:%s\n", crc, hipGetErrorString(crc));
        exit(1);
    }
    *g_f = (float*) gptr;
}

// free memory on gpu
extern "C" void cu_free(void *g_d) {
   hipError_t crc = hipFree(g_d);
   if (crc) {
      printf("hipFree Error=%d:%s\n", crc, hipGetErrorString(crc));
      exit(1);
   }
}
